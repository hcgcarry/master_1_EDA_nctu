#include "hip/hip_runtime.h"
#include"op.h"

struct op *computeResult(float* maxG,int* nodeI,int* nodeJ,int height,int width);
void printMatrix(float* matrix,int height,int width);
void printMatrixNode(int* matrix,int height,int width);
__global__ void compkernel(int nodeNum,float *graph,float *Dv,int *locked,int* group,float* d_maxG,int *d_nodeI,int * d_nodeJ)
{

	int X= blockIdx.x * blockDim.x + threadIdx.x;
	int Y= blockIdx.y * blockDim.y + threadIdx.y;
	float maxG= -1e10;
	int nodeI = -1,nodeJ = -1;
	for (int i = X; i < nodeNum; i+= blockDim.x)
	{
		if (group[i] == 0 && locked[i] == 0)
		{
			for (int j = Y; j < nodeNum; j+=blockDim.y)
			{
				if (group[j] == 1 && locked[j] == 0)
				{

					float localG = Dv[i] + Dv[j] - 2 * graph[i*nodeNum+j];
					if (localG >= maxG)
					{

						maxG = localG;
						nodeI = i;
						nodeJ = j;
					}
				}
			}
		}
	}
	int resultIndex = blockDim.y*threadIdx.x+threadIdx.y;
	d_maxG[resultIndex] = maxG;
	d_nodeI[resultIndex] = nodeI;
	d_nodeJ[resultIndex] = nodeJ;
	
}

struct op* hostFE (int nodeNum,float *graph,float *Dv,int *locked,int* group)
{
	
	float * d_graph,*d_Dv,*d_maxG;
	int *d_group,*d_locked,*d_nodeI,*d_nodeJ;
	int graphSize = nodeNum*nodeNum*sizeof(float);
	int DvSize = nodeNum*sizeof(float);
	int lockedSize = nodeNum*sizeof(float);
	int groupSize = nodeNum*sizeof(float);

	int block_size_x = 32;
	int block_size_y = 32;
	int resultSize = block_size_x*block_size_y*sizeof(float);
	float l_maxG[block_size_x*block_size_y];
	int l_nodeI[block_size_x*block_size_y];
	int l_nodeJ[block_size_x*block_size_y];
	hipMalloc((void**)&d_graph,graphSize);
	hipMalloc((void**)&d_Dv,DvSize);
	hipMalloc((void**)&d_locked,lockedSize);
	hipMalloc((void**)&d_group,groupSize);

	hipMalloc((void**)&d_maxG,resultSize);
	hipMalloc((void**)&d_nodeI,resultSize);
	hipMalloc((void**)&d_nodeJ,resultSize);

	hipMemcpy(d_graph,graph,graphSize,hipMemcpyHostToDevice);
	hipMemcpy(d_Dv,Dv,DvSize,hipMemcpyHostToDevice);
	hipMemcpy(d_locked,locked,lockedSize,hipMemcpyHostToDevice);
	hipMemcpy(d_group,group,groupSize,hipMemcpyHostToDevice);

	dim3 blockSize(block_size_x,block_size_y);

	//int group_size_x =(nodeNum % block_size_x)? nodeNum / block_size_x +1:nodeNum/block_size_x;
	//int group_size_y  = group_size_x;
	int group_size_x =1;
	int group_size_y  = 1;
	dim3 numBlock(group_size_x,group_size_y);
	compkernel<<<numBlock,blockSize>>>(nodeNum,d_graph,d_Dv,d_locked,d_group,d_maxG,d_nodeI,d_nodeJ);
	
	hipMemcpy(l_maxG,d_maxG,resultSize,hipMemcpyDeviceToHost);
	hipMemcpy(l_nodeI,d_nodeI,resultSize,hipMemcpyDeviceToHost);
	hipMemcpy(l_nodeJ,d_nodeJ,resultSize,hipMemcpyDeviceToHost);

	#ifdef debug
	cout << "gpu result l_maxG----" << endl;
	printMatrix(l_maxG,block_size_x,block_size_y);
	cout << "gpu result nodeI----" << endl;
	printMatrixNode(l_nodeI,block_size_x,block_size_y);
	cout << "gpu result nodeJ----" << endl;
	printMatrixNode(l_nodeJ,block_size_x,block_size_y);
	#endif

	hipFree(d_graph);
	hipFree(d_Dv);
	hipFree(d_locked);
	hipFree(d_group);
	hipFree(d_maxG);
	hipFree(d_nodeI);
	hipFree(d_nodeJ);
	return computeResult(l_maxG,l_nodeI,l_nodeJ,block_size_x,block_size_y);
}
void printMatrix(float* matrix,int height,int width){
	#ifdef debug
	for(int i=0;i<height;i++){
		for(int j=0;j<width;j++){
			cout << matrix[i*width+j] << " ";
		}
		cout << endl;
	}
	cout << endl;
	#endif
}
void printMatrixNode(int* matrix,int height,int width){
	#ifdef debug
	for(int i=0;i<height;i++){
		for(int j=0;j<width;j++){
			cout << matrix[i*width+j] << " ";
		}
		cout << endl;
	}
	cout << endl;
	#endif
}
struct op* computeResult(float* maxG,int * nodeI,int* nodeJ,int height,int width){
	float LmaxG = *maxG;
	int LnodeI = *nodeI, LnodeJ = *nodeJ;
	for (int i = 0; i < height*width; i++)
	{
		if (maxG[i]>= LmaxG)
		{
			LmaxG = maxG[i];
			LnodeI = nodeI[i];
			LnodeJ = nodeJ[i];
		}
	}
	struct op* result = (struct op*)malloc(sizeof(struct op));
	result->g= LmaxG;
	result->nodeI = LnodeI;
	result->nodeJ = LnodeJ;
	return result;
}

